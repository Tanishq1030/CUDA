
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 4

__global__ void matrixAdd(int A[N][N], int B[N][N], int C[N][N]) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        C[row][col] = A[row][col] + B[row][col];
    }
}

int main() {
    int A[N][N], B[N][N], C[N][N];
    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++) {
            A[i][j] = i + j;
            B[i][j] = i - j;
        }

    int (*d_A)[N], (*d_B)[N], (*d_C)[N];
    hipMalloc((void**)&d_A, N * N * sizeof(int));
    hipMalloc((void**)&d_B, N * N * sizeof(int));
    hipMalloc((void**)&d_C, N * N * sizeof(int));

    hipMemcpy(d_A, A, N*N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N*N*sizeof(int), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(2, 2);
    dim3 numBlocks(N / 2, N / 2);
    matrixAdd<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C);

    hipMemcpy(C, d_C, N*N*sizeof(int), hipMemcpyDeviceToHost);

    printf("Result matrix:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d ", C[i][j]);
        }
        printf("\n");
    }

    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    return 0;
}
