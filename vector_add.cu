#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA kernel
__global__ void vectorAdd(float *a, float *b, float *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) c[i] = a[i] + b[i];
}

int main() {
    int N = 1 << 16;  
    size_t size = N * sizeof(float);

    float *h_a = (float*)malloc(size);
    float *h_b = (float*)malloc(size);
    float *h_c = (float*)malloc(size);

    for (int i = 0; i < N; i++) {
        h_a[i] = 1.0f;
        h_b[i] = 2.0f;
    }

    float *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);

    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < 5; i++) {
        printf("h_c[%d] = %f\n", i, h_c[i]);
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
